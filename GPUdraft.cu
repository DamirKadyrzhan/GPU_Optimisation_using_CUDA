#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <omp.h>

#define M_PI 3.14276 // PI
#define c 299792458 // speed of light in a vacuum
#define mu0 M_PI*4e-7 // magnetic permeability in a vacuum 
#define eta0 c*mu0 // wave impedance in free space 

using namespace std;

double** declare_array2D(int, int); // declare all the points in 2 Dimensions

 // Source 
__global__ void tlmSource(int* Ein[], double* V1, double* V2, double* V3, double* V4, double E0, int N) 
{

    auto index = Ein[0] + Ein[1] * N;
    V1[index] = V1[index] + E0;
    V2[index] = V2[index] - E0;
    V3[index] = V3[index] - E0;
    V4[index] = V4[index] + E0;
}

// Scatter 
__global__ void tlmScatter(int NX, int NY, double* V1, double* V2, double* V3, double* V4, double Z) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < NX && y < NY) {
        double I = (2 * V1[x * NY + y] + 2 * V4[x * NY + y] - 2 * V2[x * NY + y] - 2 * V3[x * NY + y]) / (4 * Z);

        double V = 2 * V1[x * NY + y] - I * Z;         //port1
        V1[x * NY + y] = V - V1[x * NY + y];
        V = 2 * V2[x * NY + y] + I * Z;         //port2
        V2[x * NY + y] = V - V2[x * NY + y];
        V = 2 * V3[x * NY + y] + I * Z;         //port3
        V3[x * NY + y] = V - V3[x * NY + y];
        V = 2 * V4[x * NY + y] - I * Z;         //port4
        V4[x * NY + y] = V - V4[x * NY + y];
    }
}


__global__ void tlmConnect(int NX, int NY, double* V1, double* V2, double* V3, double* V4, double rXmin, double rXmax, double rYmin, double rYmax) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < NX && y < NY) {
        //connect
        if (x > 0 && y < NY - 1) {
            double tempV = V2[x * NY + y];
            V2[x * NY + y] = V4[(x - 1) * NY + y];
            V4[(x - 1) * NY + y] = tempV;
        }
        if (x < NX - 1 && y > 0) {
            double tempV = V1[x * NY + y];
            V1[x * NY + y] = V3[x * NY + (y - 1)];
            V3[x * NY + (y - 1)] = tempV;
        }
        //boundary
        if (x < NX && y == NY - 1) {
            V3[x * NY + y] = rYmax * V3[x * NY + y];
        }
        if (x < NX && y == 0) {
            V1[x * NY + y] = rYmin * V1[x * NY + y];
        }
        if (x == NX - 1 && y < NY) {
            V4[x * NY + y] = rXmax * V4[x * NY + y];
        }
        if (x == 0 && y < NY) {
            V2[x * NY + y] = rXmin * V2[x * NY + y];
        }
    }
}

__global__ void tlmApplyProbe(double out[], double* V2, double* V4, int n, int N) {
    int Eout[] = { 15,15 };
    auto index = Eout[0] + Eout[1] * N;
    out[n] = V2[index] + V4[index];
}


int main()
{

    // Variable Declarations 
    std::clock_t start = std::clock();
    int NX = 100; // number of nodes horizontally 
    int NY = 100; // number of nodes vertically 
    int NT = 100; // number of time steps 
    double dl = 1; // node line segment length
    double dt = dl / (sqrt(2.) * c); // set time step duration


    //2D mesh variables
    double I = 0, tempV = 0, E0 = 0, V = 0;
    double** V1 = declare_array2D(NX, NY);
    double** V2 = declare_array2D(NX, NY);
    double** V3 = declare_array2D(NX, NY);
    double** V4 = declare_array2D(NX, NY);

    double Z = eta0 / sqrt(2.);

    //boundary coefficients
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;

    double width = 20 * dt * sqrt(2.); // gaussian width
    double delay = 100 * dt * sqrt(2.); // set time delay before starting
    int Ein[] = { 10,10 };
    int Eout[] = { 15,15 };

    ofstream output("output.out");

    // allocate memory on the GPU
    double* d_V1;
    double* d_V2;
    double* d_V3;
    double* d_V4;
    double* v_output = (double*)malloc(NT * sizeof(double));
    for (int n = 0; n < NT; n++) {
        v_output[n] = 0;
    }
    double* d_output;

    hipMalloc((void**)&d_V1, NX * NY * sizeof(double));
    hipMalloc((void**)&d_V2, NX * NY * sizeof(double));
    hipMalloc((void**)&d_V3, NX * NY * sizeof(double));
    hipMalloc((void**)&d_V4, NX * NY * sizeof(double));
    hipMalloc((void**)&d_output, sizeof(double) * NT);

    hipMemcpy(d_V1, V1, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2, V2, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V3, V3, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V4, V4, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_output, v_output, NT * sizeof(double), hipMemcpyHostToDevice);

    // set the grid and block sizes for the kernel
    int blockSize = 16;
    dim3 block(blockSize, blockSize);
    dim3 grid((NX + blockSize - 1) / blockSize, (NY + blockSize - 1) / blockSize);



    for (int n = 0; n < NT; n++) {

        E0 = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));

        tlmSource << < 1, 1 >> > (d_V1, d_V2, d_V3, d_V4, E0, NX);
        hipDeviceSynchronize();
        tlmScatter << < grid, block >> > (NX, NY, d_V1, d_V2, d_V3, d_V4, Z);
        hipDeviceSynchronize();
        tlmConnect << < grid, block >> > (NX, NY, d_V1, d_V2, d_V3, d_V4, rXmin, rXmax, rYmin, rYmax);
        hipDeviceSynchronize();
        tlmApplyProbe << <1, 1 >> > (d_output, d_V2, d_V4, n, NX);
        hipDeviceSynchronize();

    }

    hipDeviceSynchronize();

    hipMemcpy(V1, d_V1, NX * NY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(V2, d_V2, NX * NY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(V3, d_V3, NX * NY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(V4, d_V4, NX * NY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(v_output, d_output, sizeof(double) * NT, hipMemcpyDeviceToHost);

    for (int n = 0; n < NT; n++) {
        output << n * dt << "  " << v_output[n] << endl;
    }

    hipFree(d_V1);
    hipFree(d_V2);
    hipFree(d_V3);
    hipFree(d_V4);
    hipFree(d_output);

    output.close();
    cout << "Done";
    std::cout << ((std::clock() - start) / (double)CLOCKS_PER_SEC) << '\n';
    cin.get();

    return 0;
}




double** declare_array2D(int NX, int NY) {
    double** V = new double* [NX];
    for (int x = 0; x < NX; x++) {
        V[x] = new double[NY];
    }

    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            V[x][y] = 0;
        }
    }
    return V;
}